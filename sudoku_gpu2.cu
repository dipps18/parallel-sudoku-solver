#include "hip/hip_runtime.h"
#include<iostream>
#include<stdio.h>
#include<cstdlib>
#include<algorithm>
#include<thrust/swap.h>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

const int grid_width = 81;

typedef int grid[grid_width];

void display_grid(int *h_grid)
{
    for(int i = 0; i < 9; i++)
    {
        for(int j = 0; j < 9; j++)
            std::cout << h_grid[9*i + j] << " ";
        std::cout << std::endl;
    }
}

__device__ bool is_safe(grid h_grid, int val, int row, int col)
{
    for(int i = 0; i < 9; i++)
    {
        if(i != row && h_grid[9*i + col] == val) 
            return false;
        if(i != col && h_grid[row*9 + i] == val)
            return false;
    }

    int start_row = row - row % 3;
    int start_col = col - col % 3;
    for(int i = start_row; i < start_row + 3; i++)
    {
        for(int j = start_col; j < start_col + 3; j++)
        {
            if((i != row || j != col) && h_grid[9*i + j] == val)
                return false;
        }
    }
    return true;
}

__global__ void initNewGrids(grid *curGrids, int r, int c, int curGridsSize, int* startPosition)
{
    int tid = blockDim.x*blockIdx.x + threadIdx.x;
    grid curGrid;
    if(tid < curGridsSize)
        memcpy(curGrid, curGrids[tid], sizeof(grid));
    __syncthreads();
    if(tid < curGridsSize)
    {
        int idx = startPosition[tid];
        for(int i = 1; i <= 9; i++)
        {
            if(is_safe(curGrid, i, r, c))
            {
                curGrid[r*9 + c] = i;
                memcpy(curGrids[idx++], curGrid, sizeof(grid));
            }
        }
    }
}

__global__ void possibleGrids(grid* grids, int row, int col, int curGridCount, int* startPositions)
{
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    int count = 0; //new grids that will be formed from the current grids
    if(tid < curGridCount)
    {
        grid localGrid;
        memcpy(localGrid, grids[tid], sizeof(grid));
        int mp[10] = {0};
        for(int i = 0; i < 9; i++)
        {
            int row_cell = localGrid[9*i + col];
            int col_cell = localGrid[9*row + i];
            mp[row_cell]++;
            mp[col_cell]++;
        }
        int start_row = row - row % 3;
        int start_col = col - col % 3;
        for(int i = start_row; i < start_row + 3; i++)
        {
            for(int j = start_col; j < start_col + 3; j++)
            {
                int cell = localGrid[9*i + j];
                mp[cell]++;
            }
        }
        for(int i = 1; i < 10; i++)
        {
            if(mp[i] == 0)
                count++;
        }
        startPositions[tid] = count;
    }
    
}

void solve(grid &initGrid)
{
    int threadsPerBlock = 128;
    grid *curGrids;
    int *curCount;
    int *startPositions;
    checkCudaErrors(hipMallocManaged(&curCount, sizeof(int)));
    checkCudaErrors(hipMallocManaged(&curGrids, sizeof(grid) * 20000000));
    checkCudaErrors(hipMallocManaged(&startPositions, sizeof(int) * 2000000));
    *curCount = 1;
    memcpy(curGrids, initGrid, sizeof(grid));
    for(int i = 0; i < 9; i++)
    {
        for(int j = 0; j < 9; j++)
        {
            if(initGrid[9*i + j] == 0)
            {
                int numBlocks = (*curCount + threadsPerBlock - 1)/threadsPerBlock;
                possibleGrids<<<numBlocks, threadsPerBlock>>>(curGrids, i, j, *curCount, startPositions);
                hipDeviceSynchronize();
                int prefixSum = startPositions[0];
                for(int k = 1; k < *curCount; k++) 
                {
                    int temp = startPositions[k];
                    startPositions[k] = prefixSum;
                    prefixSum += temp;
                }
                startPositions[0] = 0;
                initNewGrids<<<numBlocks, threadsPerBlock>>>(curGrids, i, j, *curCount, startPositions);
                hipDeviceSynchronize();
                *curCount =  prefixSum;
            }
        }
    }
    if(*curCount > 1) std::cout <<"\n\nERROR!\n\n";
    else
        std::cout << "\n\ngrid solved!\n\n";
    display_grid(curGrids[0]);
    checkCudaErrors(hipFree(curGrids));
    checkCudaErrors(hipFree(curCount));
    checkCudaErrors(hipFree(startPositions));
}


int main(void)
{
    //grid myGrid = {3, 0, 6, 5, 0, 8, 4, 0, 0, 5, 2, 0, 0, 0, 0, 0, 0, 0, 0, 8, 7, 0, 0, 0, 0, 3, 1, 0, 0, 3, 0, 1, 0, 0, 8, 0, 9, 0, 0, 8, 6, 3, 0, 0, 5, 0, 5, 0, 0, 9, 0, 6, 0, 0, 1, 3, 0, 0, 0, 0, 2, 5, 0 ,0, 0, 0, 0, 0, 0, 0, 7, 4, 0, 0, 5, 2, 0, 6, 3, 0, 0};
    grid myGrid2 = {5,8,6,0,7,0,0,0,0,0,0,0,9,0,1,6,0,0,0,0,0,6,0,0,0,0,0,0,0,7,0,0,0,0,0,0,9,0,2,0,1,0,3,0,5,0,0,5,0,9,0,0,0,0,0,9,0,0,4,0,0,0,8,0,0,3,5,0,0,0,0,6,0,0,0,0,2,0,4,7,0};
    //grid myGrid3 = {0,7,0000043040009610800634900094052000358460020000800530080070091902100005007040802}
    //display_grid(myGrid);
    //solve(myGrid);
    //std::cout<<"\n\n\n";
    display_grid(myGrid2);

    solve(myGrid2);
}